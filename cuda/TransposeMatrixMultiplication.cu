#include "hip/hip_runtime.h"

%%cu


#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<math.h>
#include <stdlib.h>

void printMatrix(int * matrix,int n);

void diaganalize(int* matrix1,int* matrix2,int n);

void transposeMult(int* A,int* B, int* C,int n);

void diTransposeMult(int* PA,int* PB, int* PC,int n);

int check(int* matrix1,int* matrix2,int n);

__global__ void tranposemult(int *A_dev,int* B_dev,int* tranMult_dev, int n){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i=index/n;
    int j= index%n;
    tranMult_dev[index]=0;
    if(index<n*n){
      tranMult_dev[i*n+j]=0;
      for(int k=0;k<n;k++)
      {
        tranMult_dev[i*n+j]+=A_dev[i*n+k]*B_dev[j*n+k];
     }
  }
}
__global__ void diTranposeMult(int *PA_dev,int* PB_dev,int* diTranMult_dev, int n){
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    if (t < n*n){
      int k = t/n;
      int x = t %n;
      for(int i = 0; i < n; ++i){
        int a = (n-k+i)%n;
        if( a < 0)
          a = n -a;
        a = a*n+(x+k) %n;
        diTranMult_dev[t] += PA_dev[i*n+x] *PB_dev[a];
    }
  }
}
int main(){
    int n=16;
        int blockNumber = ceil((double)n * n/ 32);
        int* A_dev;
        int* B_dev;
        int* PA_dev;
        int* PB_dev;
        int* tranMult_dev;
        int* diTranMult_dev;

        srand(time(NULL));
        int random;

        //create matrixes
        int *A = (int*)malloc(n *n* sizeof(int*));
        for(int i=0;i<n*n;i++){
            random = rand() % 3;
            A[i]=random;
        }
        hipMalloc((void**) &A_dev, n*n*sizeof(int));
        hipMemcpy(A_dev, A, n * n * sizeof(int), hipMemcpyHostToDevice);
        
        int *B = (int*)malloc(n *n* sizeof(int*));
        for(int i=0;i<n*n;i++){
            random = rand() % 3;
            B[i]=random;
        }
        hipMalloc((void**) &B_dev, n*n*sizeof(int));
        hipMemcpy(B_dev, B, n * n * sizeof(int), hipMemcpyHostToDevice);
        
        int *P_A = (int*)malloc(n *n* sizeof(int*));
        for(int i=0;i<n*n;i++){
            P_A[i]=0;
        }
        hipMalloc((void**) &PA_dev, n*n*sizeof(int));
        hipMemcpy(PA_dev, P_A, n * n * sizeof(int), hipMemcpyHostToDevice);

        int *P_B = (int*)malloc(n *n* sizeof(int*));
        for(int i=0;i<n*n;i++){
            P_B[i]=0;
        }
        hipMalloc((void**) &PB_dev, n*n*sizeof(int));
        hipMemcpy(PB_dev, P_B, n * n * sizeof(int), hipMemcpyHostToDevice);

        int *tranMult = (int*)malloc(n *n* sizeof(int*));
        for(int i=0;i<n*n;i++){
            tranMult[i]=0;
        }
        hipMalloc((void**) &tranMult_dev, n*n*sizeof(int));
        hipMemcpy(tranMult_dev, tranMult, n * n * sizeof(int), hipMemcpyHostToDevice);

        int *diTranMult = (int*)malloc(n *n* sizeof(int*));
        for(int i=0;i<n*n;i++){
            diTranMult[i]=0;
        }
        hipMalloc((void**) &diTranMult_dev, n*n*sizeof(int));
        hipMemcpy(diTranMult_dev, diTranMult, n * n * sizeof(int), hipMemcpyHostToDevice);

        //diaganalize
        diaganalize(B,P_B,n);
        diaganalize(A,P_A,n);
        hipMemcpy(PB_dev,P_B, n * n * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(PA_dev, P_A, n * n * sizeof(int), hipMemcpyHostToDevice);

        tranposemult<<<blockNumber,32>>>(A_dev,B_dev,tranMult_dev,n);
        hipMemcpy(tranMult,tranMult_dev, n * n * sizeof(int), hipMemcpyDeviceToHost);

        diTranposeMult<<<blockNumber,32>>>(PA_dev,PB_dev,diTranMult_dev,n);
        hipMemcpy(diTranMult,diTranMult_dev, n * n * sizeof(int), hipMemcpyDeviceToHost);

          //checking
        int *checkTran = (int*)malloc(n *n* sizeof(int*));
        for(int i=0;i<n*n;i++){
            checkTran[i]=0;
        }

        transposeMult(A,B,checkTran,n);
        
        int *diCheckTran = (int*)malloc(n *n* sizeof(int*));
        for(int i=0;i<n*n;i++){
            diCheckTran[i]=0;
        }

        diTransposeMult(P_A,P_B,diCheckTran,n);

        if((check(tranMult,checkTran,n)==1)&&(check(diTranMult,diCheckTran,n)==1)){
          printf("Correct\n");
      //print matrixes
          //printf("matrix A\n");
          //printMatrix(A,n);
          printf("matrix A diaganalized\n");
          printMatrix(P_A,n);
          //printf("matrix B\n");
          //printMatrix(B,n);
          printf("matrix B diaganalized\n");
          printMatrix(P_B,n);
          printf("A*B^T Matrix\n");
          printMatrix(tranMult,n);
          printf("P_A*P_B^T Matrix\n");
          printMatrix(diTranMult,n);
        }
        else{
            printf("error");
        }

      //free matrixes
        hipFree(A_dev);
        hipFree(B_dev);
        hipFree(PA_dev);
        hipFree(PB_dev);
        hipFree(tranMult_dev);
        hipFree(diTranMult_dev);
        free(A);
        free(B);
        free(P_A);
        free(P_B);
        free(tranMult);
        free(diTranMult);
        free(checkTran);
        free(diCheckTran);

    return 0;
}

void printMatrix(int* matrix,int n){
    for (int i = 0; i < n*n; i++){
        if(i%n==0){
          printf("\n");
           printf("%d ",matrix[i]);
        }
        else{
          printf("%d ",matrix[i]);
        }
    }
    printf("\n\n");
}

void diaganalize(int* matrix1,int* matrix2,int n){
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            matrix2[i*n+j]=matrix1[j*n+(i+j)%n];
        }
    }
}

int check(int* matrix1,int* matrix2,int n){
    for (int i = 0; i < n*n; ++i)
    {
        if(matrix2[i]!=matrix1[i]){
            return -1;
        }
     
    }
    return 1;
}
void transposeMult(int* A,int* B, int* C,int n){
    int *temp = (int*)malloc(n *n* sizeof(int*));
    for(int i=0;i<n*n;i++){
        temp[i]=0;
      }
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            temp[i*n+j]=B[j*n+i];
        }
    }
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            for(int k=0;k<n;k++){
                C[i*n+j]+=A[i*n+k]*temp[k*n+j];
            }
        }
    }
    free(temp);
}


void diTransposeMult(int* PA,int* PB, int* PC,int n){
    int *temp = (int*)malloc(n *n* sizeof(int*));
    for(int i=0;i<n*n;i++){
        temp[i]=0;
      }

     for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            temp[i*n+j]=PB[j*n+i];
        }
    } 
    for(int k=0;k<n;k++){
        for(int x=0;x<n;x++){
            for(int i=0;i<n;i++){
                int a = (n-k+i)%n;
                if( a < 0)
                  a = n -a;
                a = ((x+k)%n)*n+a;
                PC[k*n+x] += PA[i*n+x] *temp[a];
            }
        }
    }
    free(temp);
}
