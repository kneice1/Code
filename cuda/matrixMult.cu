#include "hip/hip_runtime.h"

%%cu


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<math.h>
#include <stdlib.h>

void multiply(int* matrix1,int* matrix2, int* matrix3,int n);

void printMatrix(int * matrix,int n);

int check(int* matrix1,int* matrix2,int n);

__global__ void matrixMultiplication(int *A_dev,int* B_dev,int * C_dev,int m, int n)
{
  int i = (blockIdx.x * blockDim.x + threadIdx.x)*m;
  int startpoint=i;
  int endpoint = startpoint+m;
 
  int y=i/n;
  int x=i%n;
 //if m is greater than 1
  if(m>1){
     if(x<n){
      if(y<n){
        for(int j=startpoint;j<endpoint;j++){
          for(int k=0;k<n;k++){
            C_dev[(y*n+x)]+= A_dev[(y*n+k)]*B_dev[(k*n+x)];
          }
          i++;
          x=i%n;
          y=i/n;
        }
      }
    }
  }
 //if m is equal to 1
 else{
      for(int k=0;k<n;k++){
          C_dev[(y*n+x)]+= A_dev[(y*n+k)]*B_dev[(k*n+x)];
      }
    }
 }

int main(){
    int n=64;
    int m=8;
    if((n*n)>m){
        int blockNumber = ceil((double)n * n/ 32*m);
        int* A_dev;
        int* B_dev;
        int* C_dev;
        srand(time(NULL));
        int random;

        //create matrices

        int *A = (int*)malloc(n *n* sizeof(int*));
        for(int i=0;i<n*n;i++){
            random = rand() % 3;
            A[i]=random;
        }
        hipMalloc((void**) &A_dev, n*n*sizeof(int));
        hipMemcpy(A_dev, A, n * n * sizeof(int), hipMemcpyHostToDevice);
        
        int *B = (int*)malloc(n *n* sizeof(int*));
        for(int i=0;i<n*n;i++){
            random = rand() % 3;
            B[i]=random;
        }
        hipMalloc((void**) &B_dev, n*n*sizeof(int));
        hipMemcpy(B_dev, B, n * n * sizeof(int), hipMemcpyHostToDevice);
        
//make zero matrices

        int *C = (int*)malloc(n *n* sizeof(int*));
        for(int i=0;i<n*n;i++){
            C[i]=0;
        }
        hipMalloc((void**) &C_dev, n*n*sizeof(int));
        hipMemcpy(C_dev, C, n * n * sizeof(int), hipMemcpyHostToDevice);

        int *D = (int*)malloc(n *n* sizeof(int*));
        for(int i=0;i<n*n;i++){
            D[i]=0;
        }

        //cuda matrix multiply
        //finds length of time to cuda multiply.
        clock_t start = clock();
        matrixMultiplication<<<blockNumber,32>> >(A_dev, B_dev,C_dev,m,n);
        clock_t end = clock();
        
        double cudaTime=(double)(end - start) / CLOCKS_PER_SEC;
        
        hipMemcpy(C, C_dev, n * n * sizeof(int), hipMemcpyDeviceToHost);

        //normal matrix multiply
        start = clock();
        multiply(A,B,D,n);
        end = clock();
        double cTime= (double)(end - start) / CLOCKS_PER_SEC;
        

        int boolean=check(C,D,n);
        if(boolean==-1){
            printf("Error");
        }
        else{
            printf("Correct\n");
            printf("C matrix multiplication time is: %f seconds\n", cTime);
            printf("Cuda matrix multiplication time is: %f seconds\n", cudaTime);
          //print matrixes
          //printMatrix(A,n);
          //printMatrix(B,n);
          //printMatrix(C,n);
          //printMatrix(D,n);
        }

        //free matrixes
        hipFree(A_dev);
        hipFree(B_dev);
        hipFree(C_dev);
        free(A);
        free(B);
        free(C);
        free(D);

    return 0;
    }
}

void printMatrix(int* matrix,int n){
    for (int i = 0; i < n*n; i++){
        if(i%n==0){
          printf("\n");
           printf("%d ",matrix[i]);
        }
        else{
          printf("%d ",matrix[i]);
        }
    }
    printf("\n\n");
}

void multiply(int* matrix1,int* matrix2, int* matrix3,int n){
    for(int i=0;i<n;i++){
        for(int j = 0; j<n; j++)
          for(int k=0;k<n;k++){
            matrix3[i*n+j]+=matrix1[i*n+k]*matrix2[k*n+j];
        }
    }
}
int check(int* matrix1,int* matrix2,int n){
    for(int i=0;i<n*n;i++){
        if(matrix1[i]!=matrix2[i]){
            return -1;
        }
    }
    return 1;
}

